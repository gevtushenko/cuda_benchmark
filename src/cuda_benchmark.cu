#include "hip/hip_runtime.h"
//
// Created by egi on 1/3/20.
//

#include "cuda_benchmark.h"

#include "fmt/format.h"
#include "fmt/color.h"
#include "fmt/core.h"
#include "../external/fmt/include/fmt/color.h"
#include "../include/cuda_benchmark.h"

namespace cuda_benchmark
{

/**
 *
 * @param clk Elapsed clocks
 * @param peak_clk in KHz
 * @return time in ms
 */
static float clk_to_t (unsigned long long int clk, int peak_clk)
{
  return (static_cast<float> (clk) / static_cast<float> (peak_clk)) * 1000000.0f;
}

controller::controller (int block_size, int gpu_id_arg)
    : gpu_id (gpu_id_arg)
    , default_block_size (block_size)
{
  hipSetDevice (gpu_id);

  hipMalloc (&device_clk_begin, block_size * sizeof (unsigned long long));
  hipMalloc (&device_clk_end, block_size * sizeof (unsigned long long));
  hipMalloc (&device_iterations, block_size * sizeof (unsigned long long));

  host_clk_begin = std::make_unique<unsigned long long[]> (block_size);
  host_clk_end = std::make_unique<unsigned long long[]> (block_size);
  host_iterations = std::make_unique<unsigned long long[]> (block_size);
}

controller::~controller ()
{
  hipFree (device_clk_begin);
  hipFree (device_clk_end);
  hipFree (device_iterations);

  if (results.empty ())
    return;

  hipDeviceProp_t prop {};
  hipGetDeviceProperties (&prop, gpu_id);

  int peak_clk {};
  hipDeviceGetAttribute (&peak_clk, hipDeviceAttributeClockRate, gpu_id);

  fmt::print ("Run on ");
  fmt::print (fmt::fg (fmt::color::yellow_green), "{0}\n", prop.name);

  const auto longest_name_size = std::max (std::max_element (results.begin (), results.end (), [] (const result &a, const result &b) {
    return a.benchmark_name.size () < b.benchmark_name.size ();
  })->benchmark_name.size (), 20ul);
  const auto longest_clock_size = std::max (std::to_string (std::max_element (results.begin (), results.end (), [] (const result &a, const result &b) {
    return std::to_string (a.latency).size () < std::to_string (b.latency).size ();
  })->latency).size (), std::string("Latency (clk)").size ());
  const auto longest_time_size = std::max (fmt::format ("{:.2f}", clk_to_t (std::max_element (results.begin (), results.end (), [peak_clk] (const result &a, const result &b) {
    return fmt::format ("{:.2f}", clk_to_t (a.latency, peak_clk)).size () < fmt::format ("{:.2f}", clk_to_t (b.latency, peak_clk)).size ();
  })->latency, peak_clk)).size (), std::string ("Latency (ns)").size ());
  const auto longest_throughtput_size = std::max (std::to_string (std::max_element (results.begin (), results.end (), [] (const result &a, const result &b) {
    return fmt::format ("{:.6f}", a.throughput).size () < fmt::format ("{:.6f}", b.throughput).size ();
  })->throughput).size (), std::string ("Throughput (ops/clk)").size ());

  fmt::print ("{0:<{1}} {2:<{3}}    {4:<{5}}    {6:<{7}}    {8}\n",
    "Benchmark", longest_name_size,
    "Latency (ns)", longest_time_size,
    "Latency (clk)", longest_clock_size,
    "Throughput (ops/clk)", longest_throughtput_size, "Operations");
  for (const auto &result: results)
    {
      fmt::print (fmt::fg (fmt::color::green),  "{0:<{1}} ", result.benchmark_name, longest_name_size);
      fmt::print (fmt::fg (fmt::color::orange), "{0:>{1}.2f}    ", clk_to_t (result.latency, peak_clk), longest_time_size);
      fmt::print (fmt::fg (fmt::color::orange), "{0:>{1}}    ", result.latency, longest_clock_size);
      fmt::print (fmt::fg (fmt::color::orange), "{0:>{1}.6f}    ", result.throughput, longest_throughtput_size);
      fmt::print (fmt::fg (fmt::color::orange), "{0} ({1})\n", result.operations, result.operations * default_block_size);
    }
}

void controller::receive_results (size_t elements) const
{
  hipMemcpy (host_clk_begin.get (), device_clk_begin, elements * sizeof (unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy (host_clk_end.get (), device_clk_end, elements * sizeof (unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy (host_iterations.get (), device_iterations, elements * sizeof (unsigned long long), hipMemcpyDeviceToHost);
}

std::pair<unsigned long long int, unsigned long long int>
controller::get_min_begin_max_end (size_t elements) const
{
  receive_results (elements);

  const unsigned long long int min_clk_begin = *std::min_element (
      host_clk_begin.get (), host_clk_begin.get () + elements);

  const unsigned long long int max_clk_end = *std::max_element (
      host_clk_end.get (), host_clk_end.get () + elements);

  return { min_clk_begin, max_clk_end };
}

void controller::process_measurements (
    std::string &&name,
    interval_type latency_interval,
    interval_type throughput_interval)
{
  const auto [latency_begin, latency_end] = latency_interval;
  const auto [throughput_begin, throughput_end] = throughput_interval;
  const auto operations = host_iterations[0];

  const auto mean_latency = (latency_end - latency_begin) / operations;
  const auto mean_throughput =
      static_cast<float>(operations * default_block_size) / static_cast<float> (throughput_end - throughput_begin);

  results.emplace_back (std::move (name), mean_latency, mean_throughput, operations);
}

}
