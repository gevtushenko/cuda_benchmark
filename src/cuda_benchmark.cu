#include "hip/hip_runtime.h"
//
// Created by egi on 1/3/20.
//

#include "cuda_benchmark.h"

#include "fmt/format.h"
#include "fmt/color.h"
#include "fmt/core.h"
#include "../external/fmt/include/fmt/color.h"

#include <algorithm>

namespace cuda_benchmark
{

controller::~controller ()
{
  hipFree (gpu_array);

  if (results.empty ())
    return;

  hipDeviceProp_t prop {};
  hipGetDeviceProperties (&prop, gpu_id);
  fmt::print ("Run on {0}\n", prop.name);

  const auto longest_name_size = std::max (std::max_element (results.begin (), results.end (), [] (const result &a, const result &b) {
    return a.benchmark_name.size () < b.benchmark_name.size ();
  })->benchmark_name.size (), 20ul);
  const auto longest_clock_size = std::max (std::to_string (std::max_element (results.begin (), results.end (), [] (const result &a, const result &b) {
    return std::to_string (a.elapsed).size () < std::to_string (b.elapsed).size ();
  })->elapsed).size (), 10ul);

  fmt::print ("{0:<{1}} {2:<{3}} {4}\n", "Benchmark", longest_name_size, "Clocks", longest_clock_size, "Iterations");
  for (const auto &result: results)
    {
      fmt::print (fmt::fg (fmt::color::green),  "{0:<{1}} ", result.benchmark_name, longest_name_size);
      fmt::print (fmt::fg (fmt::color::orange), "{0:<{1}} ", result.elapsed, longest_clock_size);
      fmt::print (fmt::fg (fmt::color::orange), "{0}\n", result.iterations);
    }
}

}
