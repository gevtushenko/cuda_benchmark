#include "hip/hip_runtime.h"
#include "include/cuda_benchmark.h"

#define REPEAT2(x)  x x
#define REPEAT4(x)  REPEAT2(x) REPEAT2(x)
#define REPEAT8(x)  REPEAT4(x) REPEAT4(x)
#define REPEAT16(x) REPEAT8(x) REPEAT8(x)
#define REPEAT32(x) REPEAT16(x) REPEAT16(x)
#define REPEAT64(x) REPEAT32(x) REPEAT32(x)
#define REPEAT128(x) REPEAT64(x) REPEAT64(x)
#define REPEAT256(x) REPEAT128(x) REPEAT128(x)

template <typename data_type>
class add_op
{
public:
  static std::string get_name () { return "add"; }
  __device__ data_type operator() (const data_type &a, const data_type &b) const { return a + b; }
};

template<>
struct add_op<int>
{
  static std::string get_name () { return "add"; }
  __device__ int operator() (const int& a, const int& b) const { int tmp; asm volatile ("add.s32 %0, %1, %2;": "=r"(tmp):"r"(a), "r"(b)); return tmp; }
};

template<>
struct add_op<long long int>
{
  static std::string get_name () { return "add"; }
  __device__ long long int operator()(const long long int& a, const long long int& b) const { long long int tmp; asm volatile ("add.s64 %0, %1, %2;": "=l"(tmp):"l"(a), "l"(b)); return tmp; }
};

template<>
struct add_op<unsigned int>
{
  static std::string get_name () { return "add"; }
  __device__ unsigned int operator()(const unsigned int& a, const unsigned int& b) const { unsigned int tmp; asm volatile ("add.u32 %0, %1, %2;": "=r"(tmp):"r"(a), "r"(b)); return tmp; }
};

template<>
struct add_op<float>
{
  static std::string get_name () { return "add"; }
  __device__ float operator()(const float& a, const float& b) const { float tmp; asm volatile ("add.f32 %0, %1, %2;": "=f"(tmp):"f"(a), "f"(b)); return tmp; }
};

template<>
struct add_op<double>
{
  static std::string get_name () { return "add"; }
  __device__ double operator()(const double& a, const double& b) const { double tmp; asm volatile ("add.f64 %0, %1, %2;": "=d"(tmp):"d"(a), "d"(b)); return tmp; }
};

template <typename data_type>
class div_op
{
public:
  static std::string get_name () { return "div"; }
  __device__ data_type operator() (const data_type &a, const data_type &b) const { return a / b; }
};

template <typename data_type>
class mul_op
{
public:
  static std::string get_name () { return "mul"; }
  __device__ data_type operator() (const data_type &a, const data_type &b) const { return a * b; }
};

template <>
class mul_op<int>
{
public:
  static std::string get_name () { return "mul"; }
  __device__ int operator() (const int &a, const int &b) const { int tmp; asm volatile ("add.s32 %0, %1, %2;" : "=r"(tmp) : "r"(a), "r"(b)); return tmp; }
};

template <typename data_type>
class mad_op
{
public:
  static std::string get_name () { return "mad"; }
  __device__ data_type operator() (const data_type &a, const data_type &b) const { data_type tmp = a; tmp += a * b; return tmp; }
};

template <typename data_type>
class exp_op
{
public:
  static std::string get_name () { return "exp"; }
  __device__ data_type operator() (const data_type &a) const { return std::exp (a); }
};

template <typename data_type>
class fast_exp_op
{
public:
  static std::string get_name () { return "fast exp"; }
  __device__ data_type operator() (const data_type &a) const { return __expf (a); }
};

template <typename data_type>
class sin_op
{
public:
  static std::string get_name () { return "sin"; }
  __device__ data_type operator() (const data_type &a) const { return std::sin (a); }
};

template <typename data_type>
class fast_sin_op
{
public:
  static std::string get_name () { return "fast sin"; }
  __device__ data_type operator() (const data_type &a) const { return __sinf (a); }
};

template <typename data_type>
std::string get_type ();

template <> std::string get_type<int> () { return "int"; }
template <> std::string get_type<float> () { return "float"; }
template <> std::string get_type<double> () { return "double"; }

template <typename data_type, typename operation_type>
void operation_benchmark_1 (cuda_benchmark::controller &controller)
{
  data_type *in {};
  const int block_size = controller.get_block_size ();
  hipMalloc (&in, block_size * sizeof (data_type));
  hipMemset (in, block_size * sizeof (data_type), 0);

  operation_type op;

  controller.benchmark (get_type<data_type> () + " " + operation_type::get_name (), [=] __device__ (cuda_benchmark::state &state)
  {
    data_type a = in[threadIdx.x];

    for (auto _ : state)
      {
        REPEAT32(a = op (a););
      }
    state.set_operations_processed (state.max_iterations () * 32);

    in[0] = a;
  });

  hipFree (in);
}

template <typename data_type, typename operation_type>
void operation_benchmark_2 (cuda_benchmark::controller &controller)
{
  data_type *in {};
  const int block_size = controller.get_block_size ();
  hipMalloc (&in, (block_size + 1) * sizeof (data_type));
  hipMemset (in, (block_size + 1) * sizeof (data_type), 0);

  operation_type op;

  controller.benchmark (get_type<data_type> () + " " + operation_type::get_name (), [=] __device__ (cuda_benchmark::state &state)
  {
    data_type a = in[threadIdx.x];
    data_type b = in[threadIdx.x + 1];

    for (auto _ : state)
      {
        REPEAT32(a = op (a, b););
      }
    state.set_operations_processed (state.max_iterations () * 32);

    in[0] = (a + b);
  });

  hipFree (in);
}

template <template <typename> typename op_type>
void operation_benchmark (cuda_benchmark::controller &controller)
{
  operation_benchmark_2<int, op_type<int>> (controller);
  operation_benchmark_2<float, op_type<float>> (controller);
  operation_benchmark_2<double, op_type<double>> (controller);
}

template <template <typename> typename op_type>
void operation_benchmark_float (cuda_benchmark::controller &controller)
{
  operation_benchmark_1<float, op_type<float>> (controller);
  operation_benchmark_1<double, op_type<double>> (controller);
}

struct node
{
public:
  node *next_node;
};

void global_access_benchmark (cuda_benchmark::controller &controller, int n, int stride)
{
  std::unique_ptr<node[]> cpu_in (new node[n]);

  node *in {};
  hipMalloc (&in, n * sizeof (node));

  for (int i = 0; i < n; i++)
    cpu_in[i].next_node = in + (i + stride) % n;
  hipMemcpy (in, cpu_in.get (), n * sizeof (node), hipMemcpyHostToDevice);

  controller.benchmark (
    "global access (stride=" + std::to_string (stride) + "; n=" + std::to_string (n) + ")",
    [=] __device__ (cuda_benchmark::state &state)
  {
    node *a = in + threadIdx.x;

    for (auto _ : state)
      {
        REPEAT32(a = a->next_node;);
      }
    state.set_operations_processed (state.max_iterations () * 32);

    __syncthreads ();
    in[0].next_node = a->next_node;
  });

  hipFree (in);
}

void divergence_benchmark (cuda_benchmark::controller &controller, int group_size)
{
  int n = 1024;

  int *in {};
  hipMalloc (&in, (n + 1) * sizeof (int));
  hipMemset (in, (n + 1) * sizeof (int), 0);

  controller.benchmark ("without divergence (group_size=" + std::to_string (group_size) + ")", [=] __device__ (cuda_benchmark::state &state) {
    int lane_id = threadIdx.x % 32;
    int group_id = lane_id / group_size;

    int a = in[threadIdx.x];
    int b = in[threadIdx.x + 1];

    for (auto _ : state)
      {
        switch (group_id)
          {
            case 0: a += b; break;
            case 1: a -= b; break;
            case 2: a ^= b; break;
            case 3: a &= b; break;
          }
      }

    in[threadIdx.x] = a;
  });

  hipFree (in);
}

void separated_pipelines_benchmark (cuda_benchmark::controller &controller)
{
  int n = 1024;

  int *in_i {};
  hipMalloc (&in_i, (n + 1) * sizeof (int));
  hipMemset (in_i, (n + 1) * sizeof (int), 0);

  float *in_f {};
  hipMalloc (&in_f, (n + 1) * sizeof (float));
  hipMemset (in_f, (n + 1) * sizeof (float), 0);

  add_op<int> op_i;
  add_op<float> op_f;

  controller.benchmark ("separated pipelines", [=] __device__ (cuda_benchmark::state &state) {
    int ai = in_i[threadIdx.x];
    int bi = in_i[threadIdx.x + 1];

    float af = in_f[threadIdx.x];
    float bf = in_f[threadIdx.x + 1];

    for (auto _ : state)
      {
        REPEAT32(ai = op_i (ai, bi); af = op_f (af, bf); );
      }
    state.set_operations_processed (state.max_iterations () * 32 * 2);

    in_i[threadIdx.x] = ai;
    in_f[threadIdx.x] = af;
  });

  hipFree (in_i);
  hipFree (in_f);
}

int main ()
{
  cuda_benchmark::controller controller;

  operation_benchmark<add_op> (controller);
  operation_benchmark<div_op> (controller);
  operation_benchmark<mul_op> (controller);
  operation_benchmark<mad_op> (controller);

  operation_benchmark_float<exp_op> (controller);
  operation_benchmark_1<float, fast_exp_op<float>> (controller);

  operation_benchmark_float<sin_op> (controller);
  operation_benchmark_1<float, fast_sin_op<float>> (controller);

  global_access_benchmark (controller, 1024, 1);
  global_access_benchmark (controller, 16 * 1024 * 1024, 4);
  global_access_benchmark (controller, 16 * 1024 * 1024, 8);

  divergence_benchmark (controller, 32);
  divergence_benchmark (controller, 16);
  divergence_benchmark (controller, 8);

  separated_pipelines_benchmark (controller);

  return 0;
}
