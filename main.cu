#include <iostream>
#include "cuda_benchmark.h"

int main ()
{
  hipSetDevice (1);

  float *in_f {};
  hipMalloc (&in_f, 2 * sizeof (float));

  float *in_d {};
  hipMalloc (&in_d, 2 * sizeof (float));

  benchmark ([=] __device__ (cuda_benchmark::state &state)
  {
    float a = in_f[threadIdx.x];
    float b = in_f[threadIdx.x + 1];

    for (auto _ : state)
      a = a + b;

    in_f[0] = (a + b);
  });

  benchmark ([=] __device__ (cuda_benchmark::state &state)
  {
    double a = in_d[threadIdx.x];
    double b = in_d[threadIdx.x + 1];

    for (auto _ : state)
      a = a + b;

    in_d[0] = (a + b);
  });

  hipFree (in_f);
  hipFree (in_d);

  return 0;
}
